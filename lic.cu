#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include "point.h"
#include <hip/hip_runtime.h>
static const int WORK_SIZE = 256;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

__host__ __device__ unsigned int bitreverse(unsigned int number) {
	number = ((0xf0f0f0f0 & number) >> 4) | ((0x0f0f0f0f & number) << 4);
	number = ((0xcccccccc & number) >> 2) | ((0x33333333 & number) << 2);
	number = ((0xaaaaaaaa & number) >> 1) | ((0x55555555 & number) << 1);
	return number;
}

/**
 * CUDA kernel function that reverses the order of bits in each element of the array.
 */
__global__ void bitreverse(void *data) {
	unsigned int *idata = (unsigned int*) data;
	idata[threadIdx.x] = bitreverse(idata[threadIdx.x]);
}

__device__ int getPixel(const Point &p, int &i, int &j, int rows, int cols)
{
  i = ((int)p.coord[0] + rows)%rows;
  j = ((int)p.coord[1] + cols)%cols;

  if (i >= 0 && i < rows && j >=0 && j < cols)
    return 1;
  return 0;
}

__device__ Vector getVector(const Point &p,   Vector *vecdata, int rows, int cols)
{
  int i,j;

  if (getPixel(p,i,j, rows, cols)) {
    return vecdata[i * cols +j];
  }
  return(Vector(0,0,0));
}

__device__ void RK(Point &p, double h, Vector *vecdata, int rows, int cols)
{
  Vector v;
  Vector k1,k2,k3,k4;

  v = getVector(p, vecdata, rows, cols);
  if (!v.iszero())
    v = v.unit();
  //v.Print();

  k1 = v*h;
  v = (getVector(p+k1*.5, vecdata, rows, cols));
  if (!v.iszero())
    v = v.unit();
  //v.Print();

  k2 = v*h;
  v = (getVector(p+k2*.5, vecdata, rows, cols));
  if (!v.iszero())
    v = v.unit();
  //v.Print();

  k3 = v*h;
  v = (getVector(p+k3, vecdata, rows, cols));
  if (!v.iszero())
    v = v.unit();
  //v.Print();

  k4 = v*h;
  p += k1/6 + k2/3 + k3/3 + k4/6;
}

__device__ void GenStreamLine(int i, int j, Point* bwd, Point* fwd, Vector *vecdata, int rows, int cols, Point* origin)
{
  Point b,f;

  *origin = f = b = Point(i+.5,j+.5);
  for (int k=0; k<M+L-1; k++) {
    RK(f,Ht, vecdata, rows, cols);
    fwd[k] = f;
    RK(b,-Ht,vecdata, rows, cols);
    bwd[k] = b;
  }
}

__device__  int validpt(Point &p, int rows,int cols) {
  int i,j;

  if (getPixel(p,i,j, rows, cols))
    return 1;
  return 0;
}

__device__ Point getSLIndex(int m, Point* bwd, Point* fwd, Point origin) {
    if (m == 0)
      return origin;
    else if (m>0)
      return fwd[m-1];
    else
      return bwd[-m-1];
}

__device__ inline double getT(Point &p, int *texdata, int rows, int cols)
{
  int i,j;

  if (getPixel(p,i,j, rows, cols))
    return texdata[i * cols + j];
  return 0;
}

__device__ double ComputeI(Point* bwd, Point* fwd, Point origin, int &numvalid, int rows, int cols, double *Idata, int *hitdata, int *texdata)
  {
    double T,k,I;
    int i,j;

    T=0;
    numvalid = 0;


    for(i=-L; i<= L; i++) {
      Point p = getSLIndex(i, bwd, fwd, origin);
      if (validpt(p, rows, cols)) {
        T += getT(p, texdata, rows, cols);
        numvalid++;
      }
    }
    if (getPixel(origin, i, j, rows, cols)) {
      k = 1./numvalid;
      // printf("GPU inside IF statement");
      Idata[i * cols + j] += I = T*k;
      // printf("[%d] = %lf", i * cols + j, T*k);
      hitdata[i * cols + j]++;
      return I;
    }
    return 0;

  }


__global__ void lic_kernel(int rows, int cols, Vector *vecdata, int *hitdata, int *texdata, double *Idata) {
  Point fwd[M+L-1];
  Point bwd[M+L-1];
  Point origin;
  int numvalid = 0;


  // GenStreamLine(0, 0, bwd, fwd, vecdata, rows, cols, &origin);
  // ComputeI(bwd, fwd, origin, numvalid, rows, cols, Idata, hitdata, texdata);
  // printf("%lf, ", Idata[0]);






  for(int i = 0; i < 10; i++) {
    for(int j = 0; j < 10; j++) {
      GenStreamLine(i, j, bwd, fwd, vecdata, rows, cols, &origin);
      ComputeI(bwd, fwd, origin, numvalid, rows, cols, Idata, hitdata, texdata);
      // printf("%d, ", i * cols + j);
      printf("%lf, ", Idata[i * cols + j]);
    }
  }

  // todo print numvalid to ensure its changed

  // int i = blockIdx.x * blockDim.x + threadIdx.x;
  // int j = blockIdx.y * blockDim.y + threadIdx.y;
  // if (hitdata[i][j] < minNumHits) {
  //   GenStreamLine(i, j, bwd, fwd, vecdata, rows, cols, &origin);
  //   ComputeI(bwd, fwd, origin, numvalid, rows, cols, Idata, hitdata, texdata);
  // }
}



void licGPU(int rows, int cols, Vector *vecdata, int *texdata) {
  Vector *vecdata_dev;
  int *hitdata_dev;
  int *texdata_dev;
  double *Idata_dev;


  CUDA_CHECK_RETURN(hipMalloc((void**) &hitdata_dev, sizeof(int) * rows * cols));
  CUDA_CHECK_RETURN(hipMalloc((void**) &vecdata_dev, sizeof(Vector) * rows * cols));
  CUDA_CHECK_RETURN(hipMalloc((void**) &texdata_dev, sizeof(int) * rows * cols));
  CUDA_CHECK_RETURN(hipMalloc((void**) &Idata_dev, sizeof(double) * rows * cols));


  // need to copy vecdata, texdata after readPts()
  CUDA_CHECK_RETURN(hipMemcpy(vecdata_dev, vecdata, sizeof(Vector) * rows * cols, hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(texdata_dev, texdata, sizeof(int) * rows * cols, hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemset(hitdata_dev, 0, sizeof(int) * rows * cols));
  CUDA_CHECK_RETURN(hipMemset(Idata_dev, 0, sizeof(double) * rows * cols));

  // set up parameters for threads structure
  dim3 dimGrid(1, 1);
  dim3 dimBlock(1, 1, 1);

  lic_kernel<<<dimGrid, dimBlock>>>(rows, cols, vecdata_dev, hitdata_dev, texdata_dev, Idata_dev);
  hipDeviceSynchronize();



  CUDA_CHECK_RETURN(hipFree((void*) hitdata_dev));
  CUDA_CHECK_RETURN(hipFree((void*) vecdata_dev));
  CUDA_CHECK_RETURN(hipFree((void*) texdata_dev));
  CUDA_CHECK_RETURN(hipFree((void*) Idata_dev));
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
/*
int main(void) {
	void *d = NULL;
	int i;
	unsigned int idata[WORK_SIZE], odata[WORK_SIZE];

	for (i = 0; i < WORK_SIZE; i++)
		idata[i] = (unsigned int) i;

	CUDA_CHECK_RETURN(hipMalloc((void**) &d, sizeof(int) * WORK_SIZE));
	CUDA_CHECK_RETURN(
			hipMemcpy(d, idata, sizeof(int) * WORK_SIZE, hipMemcpyHostToDevice));

	bitreverse<<<1, WORK_SIZE, WORK_SIZE * sizeof(int)>>>(d);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(hipMemcpy(odata, d, sizeof(int) * WORK_SIZE, hipMemcpyDeviceToHost));

	for (i = 0; i < WORK_SIZE; i++)
		printf("Input value: %u, device output: %u, host output: %u\n",
				idata[i], odata[i], bitreverse(idata[i]));

	CUDA_CHECK_RETURN(hipFree((void*) d));
	CUDA_CHECK_RETURN(hipDeviceReset());

	return 0;
}
*/
